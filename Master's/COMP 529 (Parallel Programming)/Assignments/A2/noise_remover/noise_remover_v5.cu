#include "hip/hip_runtime.h"
/*
 * noise_remover_v5.cu
 *
 * This program removes noise from an image based on Speckle Reducing Anisotropic Diffusion
 * Y. Yu, S. Acton, Speckle reducing anisotropic diffusion,
 * IEEE Transactions on Image Processing 11(11)(2002) 1260-1270 <http://people.virginia.edu/~sc5nf/01097762.pdf>
 * Original implementation is Modified by Burak BASTEM
 *
 * COMP 529 - Shukhrat Khuseynov - 0070495
 * Optimizing stat kernel (reduction) by using shared memory [optional].
 */

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define MATCH(s) (!strcmp(argv[ac], (s)))
#define TILE_DIM 32 // The block size

// returns the current time
static const double kMicro = 1.0e-6;
double get_time() {
	struct timeval TV;
	struct timezone TZ;
	const int RC = gettimeofday(&TV, &TZ);
	if(RC == -1) {
		printf("ERROR: Bad call to gettimeofday\n");
		return(-1);
	}
	return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );
}

__global__ void stat (unsigned char *image, float *sum, float *sum2, int width, int height)
{
    // making use of the shared memory for the sum reduction
    __shared__ float sumt[TILE_DIM][TILE_DIM], sum2t[TILE_DIM][TILE_DIM];

    // calculating the local indices of the element
    int idx = threadIdx.x, idy = threadIdx.y;

    // calculating the general indices of the element
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    int j = blockIdx.x*blockDim.x + threadIdx.x;

    // loading to shared memory
    if (i < height && j < width)
    {
        float tmp = image[i * width + j];
        sumt[idx][idy] = tmp;
        sum2t[idx][idy] = tmp * tmp; // --- 1 floating point arithmetic operations
    }
    else
    {
        sumt[idx][idy] = 0;
        sum2t[idx][idy] = 0;
    }

    __syncthreads();

    // doing reduction in shared memory for x axis
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (idx < s)
        {
            sumt[idx][idy] += sumt[idx + s][idy]; // --- 1 floating point arithmetic operations
            sum2t[idx][idy] += sum2t[idx + s][idy]; // --- 1 floating point arithmetic operations
        }
        __syncthreads();

    }

    if (idx == 0)
    {
        // doing reduction in shared memory for y axis
        for (unsigned int s = blockDim.y / 2; s > 0; s >>= 1)
        {
            if (idy < s)
            {
                sumt[idx][idy] += sumt[idx][idy + s]; // --- 1 floating point arithmetic operations
                sum2t[idx][idy] += sum2t[idx][idy + s]; // --- 1 floating point arithmetic operations
            }
            __syncthreads();
        }

        if (idy == 0)
        {
            atomicAdd(sum, sumt[0][0]);
            atomicAdd(sum2, sum2t[0][0]);
        }
    }
}

__global__ void compute1 (unsigned char *image, float *north_deriv, float *south_deriv, float *west_deriv, float *east_deriv, float *diff_coef, int width, int height, float std_dev)
{
    // making use of shared memory for the image
    __shared__ float imagetemp[TILE_DIM + 2][TILE_DIM + 2];

    // calculating the local indices of the element
    int idx = threadIdx.x + 1, idy = threadIdx.y + 1;

    // calculating the general indices of the element
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    int j = blockIdx.x*blockDim.x + threadIdx.x;

    // other variables
    long k;
    float gradient_square, laplacian, num, den, std_dev2;
    float imagek, north_derivk, south_derivk, west_derivk, east_derivk;

    // reading input elements into shared memory
    if (i < height && j < width)
    imagetemp[idx][idy] = image[i * width + j];

    // loading ghost cells (halos)
    if (idx == 1)
    {
        if (j>0)
        {
            imagetemp[idx - 1][idy] = image[i * width + (j - 1)];
        }

        if (j + TILE_DIM < width)
        {
            imagetemp[idx + TILE_DIM][idy] = image[i * width + (j + TILE_DIM)];
        }
    }

    if (idy == 1)
    {
        if (i>0)
        {
            imagetemp[idx][idy - 1] = image[(i - 1)* width + j];
        }

        if (i + TILE_DIM < height)
        {
            imagetemp[idx][idy + TILE_DIM] = image[(i + TILE_DIM)* width + j];
        }
    }

    __syncthreads();

    // doing the compute 1
    if ( (i>0 && i<(height-1)) && (j>0 && j<(width-1)) )
    {
        k = i * width + j; // position of current elements

        imagek = imagetemp[idx][idy];
        north_derivk = imagetemp[idx][idy - 1] - imagek; north_deriv[k] = north_derivk; // --- 1 floating point arithmetic operations
        south_derivk = imagetemp[idx][idy + 1] - imagek; south_deriv[k] = south_derivk; // --- 1 floating point arithmetic operations
        west_derivk = imagetemp[idx - 1][idy] - imagek; west_deriv[k] = west_derivk; // --- 1 floating point arithmetic operations
        east_derivk = imagetemp[idx + 1][idy] - imagek; east_deriv[k] = east_derivk; // --- 1 floating point arithmetic operations

        gradient_square = (north_derivk * north_derivk + south_derivk * south_derivk + west_derivk * west_derivk + east_derivk * east_derivk) / (imagek * imagek); // 9 floating point arithmetic operations
        laplacian = (north_derivk + south_derivk + west_derivk + east_derivk) / imagek; // 4 floating point arithmetic operations

        num = (0.5 * gradient_square) - ((1.0 / 16.0) * (laplacian * laplacian)); // 5 floating point arithmetic operations
        den = 1 + (.25 * laplacian); // 2 floating point arithmetic operations
        std_dev2 = num / (den * den); // 2 floating point arithmetic operations
        den = (std_dev2 - std_dev) / (std_dev * (1 + std_dev)); // 4 floating point arithmetic operations

        diff_coef[k] = 1.0 / (1.0 + den); // 2 floating point arithmetic operations

        if (diff_coef[k] < 0)   diff_coef[k] = 0;
        else if (diff_coef[k] > 1)  diff_coef[k] = 1;
    }
}

__global__ void compute2 (unsigned char *image, float *north_deriv, float *south_deriv, float *west_deriv, float *east_deriv, float *diff_coef, int width, int height, float lambda)
{
    // making use of shared memory for the diff_coef
    __shared__ float difftemp[TILE_DIM + 1][TILE_DIM + 1];

    // calculating the local indices of the element
    int idx = threadIdx.x, idy = threadIdx.y;

    // calculating the general indices of the element
    int i = blockIdx.y*blockDim.y + idy;
    int j = blockIdx.x*blockDim.x + idx;

    // other variables
    long k;
    float diff_coef_north, diff_coef_south, diff_coef_west, diff_coef_east;
    float divergence;

    // reading input elements into shared memory
    if (i < height && j < width)
    difftemp[idx][idy] = diff_coef[i * width + j];

    // loading ghost cells (halos)
    if (idx == 0 && (j + TILE_DIM) < width )
    {
        difftemp[idx + TILE_DIM][idy] = diff_coef[i * width + (j + TILE_DIM)];
    }

    if (idy == 0 && (i + TILE_DIM) < height )
    {
        difftemp[idx][idy + TILE_DIM] = diff_coef[(i + TILE_DIM)* width + j];
    }

    __syncthreads();

    // doing the compute 2
    if ( (i>0 && i<(height-1)) && (j>0 && j<(width-1)) )
    {
        k = i * width + j; // position of current element

        diff_coef_north = difftemp[idx][idy];
        diff_coef_south = difftemp[idx][idy + 1];
        diff_coef_west = difftemp[idx][idy];
        diff_coef_east = difftemp[idx + 1][idy];

        divergence = diff_coef_north * north_deriv[k] + diff_coef_south * south_deriv[k] + diff_coef_west * west_deriv[k] + diff_coef_east * east_deriv[k]; // --- 7 floating point arithmetic operations
        image[k] = image[k] + 0.25 * lambda * divergence; // --- 3 floating point arithmetic operations
    }

}
int main(int argc, char *argv[]) {
	// Part I: allocate and initialize variables
	double time_0, time_1, time_2, time_3, time_4, time_5, time_6, time_7, time_8;	// time variables
	time_0 = get_time();
	const char *filename = "input.pgm";
	const char *outputname = "output.png";
	int width, height, pixelWidth, n_pixels;
	int n_iter = 50;
	float lambda = 0.5;
	float mean, variance, std_dev;	//local region statistics
	float *north_deriv, *south_deriv, *west_deriv, *east_deriv;	// directional derivatives
	float sum, sum2;	// calculation variables
	float *diff_coef;	// diffusion coefficient

    // device variables
    unsigned char *d_image = NULL;
    float *d_sum = NULL, *d_sum2 = NULL;
    float *d_north_deriv = NULL, *d_south_deriv = NULL, *d_west_deriv = NULL, *d_east_deriv = NULL, *d_diff_coef = NULL;
    //int TILE_DIM2 = TILE_DIM * TILE_DIM;

	time_1 = get_time();

	// Part II: parse command line arguments
	if(argc<2) {
	  printf("Usage: %s [-i < filename>] [-iter <n_iter>] [-l <lambda>] [-o <outputfilename>]\n",argv[0]);
	  return(-1);
	}
	for(int ac=1;ac<argc;ac++) {
		if(MATCH("-i")) {
			filename = argv[++ac];
		} else if(MATCH("-iter")) {
			n_iter = atoi(argv[++ac]);
		} else if(MATCH("-l")) {
			lambda = atof(argv[++ac]);
		} else if(MATCH("-o")) {
			outputname = argv[++ac];
		} else {
		printf("Usage: %s [-i < filename>] [-iter <n_iter>] [-l <lambda>] [-o <outputfilename>]\n",argv[0]);
		return(-1);
		}
	}
	time_2 = get_time();

	// Part III: read image
	printf("Reading image...\n");
	unsigned char *image = stbi_load(filename, &width, &height, &pixelWidth, 0);
	if (!image) {
		fprintf(stderr, "Couldn't load image.\n");
		return (-1);
	}
	printf("Image Read. Width : %d, Height : %d, nComp: %d\n",width,height,pixelWidth);
	n_pixels = height * width;
	time_3 = get_time();

	// Part IV: allocate variables
	north_deriv = (float*) malloc(sizeof(float) * n_pixels);	// north derivative
	south_deriv = (float*) malloc(sizeof(float) * n_pixels);	// south derivative
	west_deriv = (float*) malloc(sizeof(float) * n_pixels);	// west derivative
	east_deriv = (float*) malloc(sizeof(float) * n_pixels);	// east derivative
	diff_coef  = (float*) malloc(sizeof(float) * n_pixels);	// diffusion coefficient

	// allocate storage for the device
    hipMalloc((void**)&d_image, sizeof(unsigned char) * n_pixels);
    hipMalloc((void**)&d_sum, sizeof(float));
    hipMalloc((void**)&d_sum2, sizeof(float));

    hipMalloc((void**)&d_north_deriv, sizeof(float) * n_pixels);
    hipMalloc((void**)&d_south_deriv, sizeof(float) * n_pixels);
    hipMalloc((void**)&d_west_deriv, sizeof(float) * n_pixels);
    hipMalloc((void**)&d_east_deriv, sizeof(float) * n_pixels);
    hipMalloc((void**)&d_diff_coef, sizeof(float) * n_pixels);

    const dim3 block(TILE_DIM,TILE_DIM);
    const dim3 grid( (width + TILE_DIM - 1) / block.x, (height + TILE_DIM - 1) / block.y);

	time_4 = get_time();

	// Part V: compute --- n_iter * (reduction + 42 * (height-1) * (width-1) + 6) floating point arithmetic operations in totaL
	// reduction --- height * width * (1 + 2*log2(TILE_DIM/2)) + 2*height * log2(TILE_DIM/2) + 2*(height * width) / (TILE_DIM * TILE_DIM)
	for (int iter = 0; iter < n_iter; iter++)
    {
        // REDUCTION AND STATISTICS

        sum = 0; sum2 = 0;

        // copying input to the device
        hipMemcpy(d_image, &image[0], sizeof(unsigned char) * n_pixels, hipMemcpyHostToDevice);
        hipMemcpy(d_sum, &sum, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_sum2, &sum2, sizeof(float), hipMemcpyHostToDevice);

        // running the stat kernel
        stat<<<grid, block>>>(d_image, d_sum, d_sum2, width, height);

        // copying output back to the host
        hipMemcpy(&sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&sum2, d_sum2, sizeof(float), hipMemcpyDeviceToHost);

		mean = sum / n_pixels; // --- 1 floating point arithmetic operations
		variance = (sum2 / n_pixels) - mean * mean; // --- 3 floating point arithmetic operations
		std_dev = variance / (mean * mean); // --- 2 floating point arithmetic operations


		//COMPUTE 1

        // copying input to the device
        hipMemcpy(d_image, &image[0], sizeof(unsigned char) * n_pixels, hipMemcpyHostToDevice);

        // running the compute1 kernel
        compute1<<<grid, block>>>(d_image, d_north_deriv, d_south_deriv, d_west_deriv, d_east_deriv, d_diff_coef, width, height, std_dev);

        // copying output back to the host
        hipMemcpy(&north_deriv[0], d_north_deriv, sizeof(float) * n_pixels, hipMemcpyDeviceToHost);
        hipMemcpy(&south_deriv[0], d_south_deriv, sizeof(float) * n_pixels, hipMemcpyDeviceToHost);
        hipMemcpy(&west_deriv[0], d_west_deriv, sizeof(float) * n_pixels, hipMemcpyDeviceToHost);
        hipMemcpy(&east_deriv[0], d_east_deriv, sizeof(float) * n_pixels, hipMemcpyDeviceToHost);
        hipMemcpy(&diff_coef[0], d_diff_coef, sizeof(float) * n_pixels, hipMemcpyDeviceToHost);


		// COMPUTE 2

        // copying input to the device
        hipMemcpy(d_image, &image[0], sizeof(unsigned char) * n_pixels, hipMemcpyHostToDevice);
        hipMemcpy(d_north_deriv, &north_deriv[0], sizeof(float) * n_pixels, hipMemcpyHostToDevice);
        hipMemcpy(d_south_deriv, &south_deriv[0], sizeof(float) * n_pixels, hipMemcpyHostToDevice);
        hipMemcpy(d_west_deriv, &west_deriv[0], sizeof(float) * n_pixels, hipMemcpyHostToDevice);
        hipMemcpy(d_east_deriv, &east_deriv[0], sizeof(float) * n_pixels, hipMemcpyHostToDevice);
        hipMemcpy(d_diff_coef, &diff_coef[0], sizeof(float) * n_pixels, hipMemcpyHostToDevice);

        // running the compute2 kernel
        compute2<<<grid, block>>>(d_image, d_north_deriv, d_south_deriv, d_west_deriv, d_east_deriv, d_diff_coef, width, height, lambda);

        // copying output back to the host
        hipMemcpy(&image[0], d_image, sizeof(unsigned char) * n_pixels, hipMemcpyDeviceToHost);
	}
	time_5 = get_time();

	// Part VI: write image to file
	stbi_write_png(outputname, width, height, pixelWidth, image, 0);
	time_6 = get_time();

	// Part VII: get average of sum of pixels for testing and calculate GFLOPS
	// FOR VALIDATION - DO NOT PARALLELIZE
	float test = 0;
	for (int i = 0; i < height; i++) {
			for (int j = 0; j < width; j++) {
				test += image[i * width + j];
		}
	}
	test /= n_pixels;

    float reduct = height * width * (1 + 2*log2((float) TILE_DIM/2)) + 2*height * log2((float) TILE_DIM/2) + 2*(height * width) / (TILE_DIM * TILE_DIM);
	float gflops = (float) (n_iter * 1E-9 * (reduct + 42 * (height-1) * (width-1) + 6)) / (time_5 - time_4);
	time_7 = get_time();

	// Part VII: deallocate variables
	stbi_image_free(image);
	free(north_deriv);
	free(south_deriv);
	free(west_deriv);
	free(east_deriv);
	free(diff_coef);

	hipFree(d_image);
	hipFree(d_sum);
	hipFree(d_sum2);

	hipFree(d_north_deriv);
	hipFree(d_south_deriv);
	hipFree(d_west_deriv);
	hipFree(d_east_deriv);
	hipFree(d_diff_coef);

	time_8 = get_time();

	// print
	printf("Time spent in different stages of the application:\n");
	printf("%9.6f s => Part I: allocate and initialize variables\n", (time_1 - time_0));
	printf("%9.6f s => Part II: parse command line arguments\n", (time_2 - time_1));
	printf("%9.6f s => Part III: read image\n", (time_3 - time_2));
	printf("%9.6f s => Part IV: allocate variables\n", (time_4 - time_3));
	printf("%9.6f s => Part V: compute\n", (time_5 - time_4));
	printf("%9.6f s => Part VI: write image to file\n", (time_6 - time_5));
	printf("%9.6f s => Part VII: get average of sum of pixels for testing and calculate GFLOPS\n", (time_7 - time_6));
	printf("%9.6f s => Part VIII: deallocate variables\n", (time_7 - time_6));
	printf("Total time: %9.6f s\n", (time_8 - time_0));
	printf("Average of sum of pixels: %9.6f\n", test);
	printf("GFLOPS: %f\n", gflops);
	return 0;
}

